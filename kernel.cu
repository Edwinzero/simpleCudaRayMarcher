#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "cutil_math.h"
#include "hiprand/hiprand_kernel.h"

#include <stdio.h>
#include <iostream>
#include <string>
#include <chrono>
#include "lodepng.h"
#include "sdf_util.hpp"

#define BLOCK_SIZE 8
#define BOUNCES 2
#define SAMPLES 2 // Total number of samples is SAMPLES*SAMPLES
#define EPS 1e-5
#define MINDIST 1.8e-3
#define PUSH MINDIST*2
#define M_PI 3.14159265359
#define FRAMES 100

// Purely random pixel sample
inline float2 __device__ getRandomSample(hiprandState* state) 
{
	return make_float2(hiprand_uniform(state), hiprand_uniform(state));
}

// Random sample in nth subpixel
inline float2 __device__ getJitteredSample(int n, hiprandState* state) {
	float2 rand_vec = make_float2(hiprand_uniform(state) * (1.0f / SAMPLES), hiprand_uniform(state) * (1.0f / SAMPLES));
	float2 result = make_float2((n % SAMPLES) * 1.0f / SAMPLES, (n / SAMPLES) * 1.0f / SAMPLES);
	return result + rand_vec;
}

float3 __device__ orient(const float3& n, hiprandState* state) 
{
	// rejection sampling hemisphere
	float x = 1.0f, y = 1.0f;

	while (x * x + y * y > 1.0f) 
	{
		x = (hiprand_uniform(state) - 0.5f) * 2.0f;
		y = (hiprand_uniform(state) - 0.5f) * 2.0f;
	}
	float z = sqrtf(1 - x * x - y * y);
	float3 in = normalize(make_float3(x, y, z));

	// Create vector that is not the same as n
	float3 absn = fabs(n);
	float3 q = n;
	if (absn.x <= absn.y && absn.x <= absn.z)  q.x = 1;
	else if (absn.y <= absn.x && absn.y <= absn.z) q.y = 1;
	else q.z = 1;

	// Basis creation, result is just a rolled out matrix multiplication of basis matrix and in vector
	float3 t = normalize(cross(n, q));
	float3 b = normalize(cross(n, t));
	return normalize(make_float3(t.x * in.x + b.x * in.y + n.x * in.z,
								 t.y * in.x + b.y * in.y + n.y * in.z,
								 t.z * in.x + b.z * in.y + n.z * in.z));
}

struct Hit 
{
	bool isHit = 0;
	float3 pos;
	float3 normal;
	float3 color;
};

struct Camera 
{
	float3 pos;
	float3 dir;
	float halffov;
	float maxdist = 10.0f;
	float3 up;
	float3 side;
};


// Distance estimation function
float __device__ DE(const float3& pos, float time) 
{
	return mandelbulbScene(pos, time);
	//return sphereScene(pos);
	//return cornellBoxScene(pos);
	//return mengerScene(pos, 6);
	//return testFractalScene(pos, time);
}

float3 __device__ sceneColor(const float3& pos, float time) 
{
	return make_float3(0.85f);
	//return mandelbulbColor(pos, time);
	//return sphereColor(pos);
	//return cornellBoxColor(pos);
}

// Ray marching function, similar to intersect function in normal ray tracers
__device__ Hit march(const float3& orig, const float3& direction, float time) 
{
	float totaldist = 0.0f;
	float maxdist = length(direction);
	float3 pos = orig; float3 dir = normalize(direction);
	float3 col = make_float3(0.85f, 0.85f, 0.85f);

	Hit hit;

	while (totaldist < maxdist) 
	{
		float t = DE(pos, time);

		// If distance is less than this then it is a hit.
		if (t < MINDIST) 
		{
			// Calculate gradient (normal)
			float fx = (DE(make_float3(pos.x + EPS, pos.y, pos.z), time) - DE(make_float3(pos.x - EPS, pos.y, pos.z), time)) / (2.0f * EPS);
			float fy = (DE(make_float3(pos.x, pos.y + EPS, pos.z), time) - DE(make_float3(pos.x, pos.y - EPS, pos.z), time)) / (2.0f * EPS);
			float fz = (DE(make_float3(pos.x, pos.y, pos.z + EPS), time) - DE(make_float3(pos.x, pos.y, pos.z - EPS), time)) / (2.0f * EPS);
			float3 normal = normalize(make_float3(fx - t, fy - t, fz - t));
			// faceforward
			if (dot(-dir, normal) < 0) normal = -normal;

			// create hit
			hit.isHit = true;
			hit.pos = pos;
			hit.normal = normal;
			hit.color = sceneColor(pos, time);
			return hit;
		}

		// step forwards by t if no hit
		totaldist += t;
		pos += t * dir;
	}

	return hit;
}

// Path tracing function
__device__ float3 trace(const float3& orig, const float3& direction, hiprandState* state, float time)
{
	float raylen = length(direction);
	float3 dir = direction;
	float3 o = orig;
	float3 p = make_float3(0.0f); float3 n = make_float3(0.0f);
	float3 mask = make_float3(1.0f); float3 color = make_float3(0.0f);

	Hit rayhit = march(o, dir, time);
	
	for (int i = 0; i < BOUNCES + 1; i++) 
	{
		if (rayhit.isHit) 
		{
			p = rayhit.pos; n = rayhit.normal;
			// Create new ray direction
			float3 d = orient(n, state);
			o = p + n * PUSH;
			mask *= rayhit.color;
			dir = raylen * d;
			// Fire new ray if there are bounces left
			if (i < BOUNCES) rayhit = march(o, dir, time);
		}
		else if (i == 0) return make_float3(0.0f); // black background
		else 
		{
			color += make_float3(1.0f) * mask; // add color when light (sky) is hit
			break;
		}
	}
	
	return color;
}

__global__ void render(int width, int height, float* result, Camera cam, unsigned long long seed, float time)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	if (x >= width || y >= height) return;

	float3 color = make_float3(0.0f);

	int block = blockIdx.x + blockIdx.y * gridDim.x;
	unsigned long long idx = block * (blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x) + threadIdx.x;
	
	hiprandState state;
	hiprand_init(idx + seed, 0, 0, &state);

	float2 samp = make_float2(x, y);
	
	for (int i = 0; i < SAMPLES * SAMPLES; i++) {
		//float2 offset = getRandomSample(&state);
		float2 offset = getJitteredSample(i, &state);
		float2 sample = samp + offset;
		float nx = (sample.x / float(width) - 0.5f) * 2.0f;
		float ny = -(sample.y / float(height) - 0.5f) * 2.0f;
		ny *= float(height) / float(width);
		float3 pt = cam.pos + cam.side * cam.halffov * nx + cam.up * ny * cam.halffov + cam.dir;
		float3 raydir = normalize(pt - cam.pos);
		color += trace(cam.pos, raydir * cam.maxdist, &state, time);
	}
	
	color /= (SAMPLES * SAMPLES);

	result[x * 3 + 3 * y * width + 0] = color.x;
	result[x * 3 + 3 * y * width + 1] = color.y;
	result[x * 3 + 3 * y * width + 2] = color.z;
}

void saveImage(std::string path, int width, int height, const float colors[]) 
{
	std::vector<unsigned char> output;
	output.resize(4 * width * height);
	for (int i = 0; i < width * height; i++)
	{
		output[i * 4 + 0] = static_cast<unsigned char>(std::fmax(std::fmin(colors[i * 3 + 0] * 255, 255), 0));
		output[i * 4 + 1] = static_cast<unsigned char>(std::fmax(std::fmin(colors[i * 3 + 1] * 255, 255), 0));
		output[i * 4 + 2] = static_cast<unsigned char>(std::fmax(std::fmin(colors[i * 3 + 2] * 255, 255), 0));
		output[i * 4 + 3] = 255;
	}
	unsigned error = lodepng::encode(path, output, width, height);
	if (error) std::cout << "An error occurred: " << lodepng_error_text(error) << std::endl;
}

int main()
{
	int width = 1920, height = 1080;
	dim3 threads(BLOCK_SIZE, BLOCK_SIZE);
	dim3 blocks(width / threads.x + 1, height / threads.y + 1);
	
	Camera cam;
	cam.pos = make_float3(-1.0f, 1.5f, -3.0f);
	//cam.pos = make_float3(0, 0.4f, -1.4f);
	cam.dir = normalize(-cam.pos);
	cam.side = normalize(cross(cam.dir, make_float3(0, 1, 0)));
	cam.up = normalize(cross(cam.side, cam.dir));
	float fov = 128.0f / 180.0f * float(M_PI);
	cam.halffov = std::tan(fov / 2.0f);

	for (int i = 0; i < FRAMES; i++) {
		float *deviceImage;
		hipMalloc(&deviceImage, 3 * width * height * sizeof(float));
		
		unsigned long long seed = std::chrono::duration_cast<std::chrono::milliseconds>(std::chrono::system_clock::now().time_since_epoch()).count();
		
		float t = 0.0f;
		if (FRAMES > 1) t = float(i) / (FRAMES - 1.0f);

		render << <blocks, threads >> >(width, height, deviceImage, cam, seed, t);
		
		float *hostImage = (float*)malloc(3 * width * height * sizeof(float));
		hipMemcpy(hostImage, deviceImage, 3 * width * height * sizeof(float), hipMemcpyDeviceToHost);
		std::string imageName = "renders/render_" + std::to_string(i + 1) + ".png";
		saveImage(imageName, width, height, hostImage);
		hipFree(deviceImage);
		free(hostImage);

		std::cout << "Frame " << (i + 1) << " done! Saved as " << imageName << "." << std::endl;
	}

	return 0;
}
